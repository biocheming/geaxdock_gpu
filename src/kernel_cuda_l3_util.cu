#include "hip/hip_runtime.h"
/*
#include <cstdlib>
#include <cstdio>

#include "dock.h"
#include "gpu.cuh"

#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
*/





__device__ void
InitAcs_d (const int bidx)
{
  if (blockIdx.x == 0) {
    for (int i = bidx; i < MAXREP; i += TperB) {
      acs_mc_dc[i] = 0;
      acs_temp_exchg_dc[i] = 0;
    }
  }
}

__device__ void
InitLigRecord_d (const int bidx, const int myreplica, const int rep_begin)
{
  for (int s2s3 = 0; s2s3 < steps_per_dump_dc; ++s2s3) {
    LigRecordSingleStep *myrecord =
      &ligrecord_dc[myreplica - rep_begin].step[s2s3];
    myrecord->replica.idx_rep = 0;
    myrecord->replica.idx_prt = 0;
    myrecord->replica.idx_tmp = 0;
    myrecord->replica.idx_lig = 0;

    for (int i = 0; i < MAXWEI; ++i)
      myrecord->energy.e[i] = 0.0f;

    for (int i = 0; i < 6; ++i)
      myrecord->movematrix[i] = 0.0f;

    myrecord->step = 0;
  }

}


/*
__forceinline__
__device__ void
BackupLigCoord_d (const int bidx, Ligand *mylig)
{

  const LigCoord *src = &mylig->coord_old;
  LigCoord *dst = &mylig->coord_bkup;

  for (int atom = bidx; atom < lna_dc; atom += TperB) {
    dst->x[atom] = src->x[atom];
    dst->y[atom] = src->y[atom];
    dst->z[atom] = src->z[atom];
  }
  if (bidx < 3)
    dst->center[bidx] = src->center[bidx];

}
*/



__device__ void
RecordLigand_d (const int bidx, const int s1, const int s2s3,
		const int myreplica, const int rep_begin,
		const Ligand * mylig, const int is_record)
{
  /*
     if (bidx == 0) // && myreplica == 0)
     printf ("rep %d, iter %d, rep_begin %d, n_rep %d, idx_rep %d\n",
     myreplica, s2, rep_begin, n_rep_dc, replica_dc[myreplica].idx_rep);

     if (myreplica == 0) {
     PrintEnergy2_d (bidx, mylig, myreplica, s1 + s2s3, 2);
     }
   */

  if (is_record == 1 && bidx == 0) {
    const int next_ptr = ligrecord_dc[myreplica - rep_begin].next_ptr;
    ligrecord_dc[myreplica - rep_begin].next_ptr = next_ptr + 1;
    LigRecordSingleStep *myrecord = &ligrecord_dc[myreplica - rep_begin].step[next_ptr];

    myrecord->replica = replica_dc[myreplica];
    myrecord->energy = mylig->energy_old;
    for (int i = 0; i < 6; ++i)
      myrecord->movematrix[i] = mylig->movematrix_old[i];
    myrecord->step = s1 + s2s3;
  }

}






__forceinline__ __device__ float
MyRand_d ()
{
  const int gidx =
    blockDim.x * blockDim.y * blockIdx.x +
    blockDim.x * threadIdx.y + threadIdx.x;
  hiprandState myseed = curandstate_dc[gidx];
  float randdd = hiprand_uniform (&myseed);
  curandstate_dc[gidx] = myseed;

  return randdd;
}




/*
__forceinline__
__device__ int
Mininal_int_d (const int a, const int b)
{
 return a < b ? a : b;
}
*/





__forceinline__ __device__ void
SumReduction1D_d (const int bidx, float *a)
{
  __syncthreads ();

  for (int stride = TperB / 2; stride >= 1; stride >>= 1) {
    if (bidx < stride)
      a[bidx] += a[stride + bidx];
    __syncthreads ();
  }
}

__forceinline__ __device__ void
SumReduction_int_1D_4_d (const int bidx, int *a, int *b, int *c, int *d)
{
  __syncthreads ();

  for (int stride = TperB / 2; stride >= 1; stride >>= 1) {
    if (bidx < stride) {
      a[bidx] += a[stride + bidx];
      b[bidx] += b[stride + bidx];
      c[bidx] += c[stride + bidx];
      d[bidx] += d[stride + bidx];
    }
    __syncthreads ();
  }
}

__forceinline__ __device__ void
SumReduction1D_5_d (const int bidx, float *a, float *b, float *c, float *d,
		    float *e)
{
  __syncthreads ();

  for (int stride = TperB / 2; stride >= 1; stride >>= 1) {
    if (bidx < stride) {
      a[bidx] += a[stride + bidx];
      b[bidx] += b[stride + bidx];
      c[bidx] += c[stride + bidx];
      d[bidx] += d[stride + bidx];
      e[bidx] += e[stride + bidx];
    }
    __syncthreads ();
  }
}


__forceinline__ __device__ void
SumReduction2D_d (float a[BDy][BDx])
{
  __syncthreads ();

  for (int stride = BDx / 2; stride >= 1; stride >>= 1) {
    if (threadIdx.x < stride) {
      a[threadIdx.y][threadIdx.x] += a[threadIdx.y][stride + threadIdx.x];
    }
    __syncthreads ();
  }
}


__forceinline__ __device__ void
SumReduction2D_2_d (float a[BDy][BDx], float b[BDy][BDx])
{
  __syncthreads ();

  for (int stride = BDx / 2; stride >= 1; stride >>= 1) {
    if (threadIdx.x < stride) {
      a[threadIdx.y][threadIdx.x] += a[threadIdx.y][stride + threadIdx.x];
      b[threadIdx.y][threadIdx.x] += b[threadIdx.y][stride + threadIdx.x];
    }
    __syncthreads ();
  }
}


__forceinline__ __device__ float
NormPdf (float x, float loc, float scale)
{

  float norm_para, prob, pdf_val;

  norm_para = 1 / (scale * sqrt (2 * PI));
  prob = exp (0.f - (x - loc) * (x - loc) / (2 * scale * scale));

  pdf_val = norm_para * prob;

  return pdf_val;
}

__forceinline__ __device__ float
CauchyPdf (float x, float loc, float scale)
{
  float norm_para, prob, pdf_val;

  norm_para = 1 / (PI * scale);
  prob = 1 / (1 + ((x - loc) / scale) * ((x - loc) / scale));

  pdf_val = norm_para * prob;

  return pdf_val;
}


__forceinline__ __device__ float
LogisticPdf (float x, float loc, float scale)
{
  float norm_para, e_power, prob, pdf_val;

  norm_para = 1 / scale;
  e_power = exp (-(x - loc) / scale);
  prob = e_power / powf (1 + e_power, 2.0);

  pdf_val = norm_para * prob;

  return pdf_val;
}

__forceinline__ __device__ float
WaldPdf (float x, float loc, float scale)
{
  float norm_para, prob, pdf_val;

  float normed_x = (x - loc) / scale;

  norm_para = 1 / (sqrt (2 * PI * powf (normed_x, 3.0)) * scale);
  prob = exp (-pow (normed_x - 1, 2) / (2 * normed_x));

  if (normed_x < 0)
    pdf_val = 0.00000001f;
  else
    pdf_val = norm_para * prob;

  return pdf_val;
}

__forceinline__ __device__ float
LaplacePdf (float x, float loc, float scale)
{
  float normed_x, pdf_val;

  normed_x = fabs (x - loc) / scale;

  pdf_val = (1 / (2 * scale)) * exp (-normed_x);

  return pdf_val;
}
